#include "hip/hip_runtime.h"
// === Copyright (c) 2020-2021 easimer.net. All rights reserved. ===
//
// Purpose: CUDA computation backend
//

#define GLM_FORCE_CUDA
#include <cassert>
#include <array>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <numeric>
#include "l_iterators.h"
#include <glm/glm.hpp>
#include "softbody.h"
#include "cuda_linalg.cuh"
#include "cuda_helper_math.h"
#define SB_BENCHMARK 1
#include "s_benchmark.h"
#include "s_compute_backend.h"
#define CUDA_SUCCEEDED(HR, APICALL) ((HR = APICALL) == hipSuccess)
#define ASSERT_CUDA_SUCCEEDED(APICALL) ((APICALL) == hipSuccess)

#define NUMBER_OF_CLUSTERS(idx) (s.edges[(idx)].size() + 1)

// TODO(danielm): double leading underscores violate the standard
#define __hybrid__ __device__ __host__

template<typename T>
struct CUDA_Array {
    CUDA_Array() : d_buf(nullptr), N(0) {
    }

    CUDA_Array(size_t N) : d_buf(nullptr), N(N) {
        if(N != 0) {
            ASSERT_CUDA_SUCCEEDED(hipMalloc(&d_buf, N * sizeof(T)));
        }
    }

    CUDA_Array(CUDA_Array const& other) : d_buf(nullptr), N(other.N) {
        if(!other.is_empty()) {
            ASSERT_CUDA_SUCCEEDED(hipMalloc(&d_buf, N * sizeof(T)));
            ASSERT_CUDA_SUCCEEDED(hipMemcpy(d_buf, other.d_buf, N * sizeof(T), hipMemcpyDeviceToDevice));
        }
    }

    CUDA_Array(CUDA_Array&& other) : d_buf(nullptr), N(0) {
        std::swap(d_buf, other.d_buf);
        std::swap(N, other.N);
    }

    ~CUDA_Array() {
        if(d_buf != nullptr) {
            ASSERT_CUDA_SUCCEEDED(hipFree(d_buf));
        }
    }

    CUDA_Array& operator=(CUDA_Array const& other) {
        if(!is_empty()) {
            ASSERT_CUDA_SUCCEEDED(hipFree(d_buf));
            d_buf = nullptr;
            N = 0;
        }

        N = other.N;
        if(N != 0) {
            ASSERT_CUDA_SUCCEEDED(hipMalloc(&d_buf, N * sizeof(T)));
            ASSERT_CUDA_SUCCEEDED(hipMemcpy(d_buf, other.d_buf, N * sizeof(T), hipMemcpyDeviceToDevice));
        }

        return *this;
    }

    CUDA_Array& operator=(CUDA_Array&& other) {
        if(!is_empty()) {
            ASSERT_CUDA_SUCCEEDED(hipFree(d_buf));
            d_buf = nullptr;
            N = 0;
        }

        std::swap(d_buf, other.d_buf);
        std::swap(N, other.N);

        return *this;
    }

    bool is_empty() const {
        return d_buf == nullptr;
    }

    size_t bytes() const {
        return N * sizeof(T);
    }

    hipError_t write_async(T const* src, hipStream_t stream) {
        return hipMemcpyAsync(d_buf, src, bytes(), hipMemcpyHostToDevice, stream);
    }

    hipError_t read_async(T* dst, hipStream_t stream) {
        return hipMemcpyAsync(dst, d_buf, bytes(), hipMemcpyDeviceToHost, stream);
    }

    hipError_t read(T* dst) {
        return hipMemcpy(dst, d_buf, bytes(), hipMemcpyDeviceToHost);
    }

    operator T*() {
        return d_buf;
    }

    T* d_buf;
    size_t N;
};

__hybrid__ float4 angle_axis(float a, float4 axis) {
    float s = sin(0.5f * a);

    float4 v = s * axis;
    float w = cos(0.5f * a);

    return make_float4(v.x, v.y, v.z, w);
}

__hybrid__ float4 quat_quat_mul(float4 p, float4 q) {
    float4 r;

    r.w = p.w * q.w - p.x * q.x - p.y * q.y - p.z * q.z;
    r.x = p.w * q.x + p.x * q.w + p.y * q.z - p.z * q.y;
    r.y = p.w * q.y - p.x * q.z + p.y * q.w + p.z * q.x;
    r.z = p.w * q.z + p.x * q.y - p.y * q.x + p.z * q.w;

    return r;
}

__hybrid__ float4 mueller_rotation_extraction_impl(
    float4 const* A,
    float4 q
) {
#define MAX_ITER (16)
    float4 t = q;
    for(int iter = 0; iter < MAX_ITER; iter++) {
        float4 R[4];
        quat_to_mat(R, t);
        float3 r0_xyz = xyz(R[0]);
        float3 r1_xyz = xyz(R[1]);
        float3 r2_xyz = xyz(R[2]);
        float3 a0_xyz = xyz(A[0]);
        float3 a1_xyz = xyz(A[1]);
        float3 a2_xyz = xyz(A[2]);
        float4 omega_v = make_float4(cross(r0_xyz, a0_xyz) + cross(r1_xyz, a1_xyz) + cross(r2_xyz, a2_xyz), 0);
        float omega_s = 1.0f / fabs(dot(r0_xyz, a0_xyz) + dot(r1_xyz, a1_xyz) + dot(r2_xyz, a2_xyz)) + 1.0e-9;
        
        float4 omega = omega_s * omega_v;
        float w = length(omega);
        if(w < 1.0e-9) {
            break;
        }

        t = normalize(quat_quat_mul(angle_axis(w, (1 / w) * omega), t));
    }

    return t;
}

__hybrid__ void calculate_A_i(
    float4* A_i,
    float mass,
    float4 orientation,
    float4 size,
    float4 predicted_position,
    float4 bind_pose,
    float4 center_of_mass,
    float4 bind_pose_center_of_mass
) {
    float4 temp[4];
    float4 diag[4];
    float4 orient[4];
    float const s = 1.0f / 5.0f;

    quat_to_mat(orient, orientation);
    diagonal3x3(diag, size * size);
    mat_mul(A_i, diag, orient);
    mat_scale(s, A_i);

    outer_product(temp, predicted_position, bind_pose);
    mat_add_assign(A_i, temp);
    outer_product(temp, center_of_mass, bind_pose_center_of_mass);
    mat_sub_assign(A_i, temp);
    mat_scale(mass, A_i);
}

__hybrid__ void calculate_cluster_moment_matrix(
    float4* A,
    unsigned i,
    float const* adjacency, unsigned N,
    float const* masses,
    float4 const* predicted_orientations,
    float4 const* sizes,
    float4 const* predicted_positions,
    float4 const* bind_pose,
    float4 const* centers_of_masses,
    float4 const* bind_pose_centers_of_masses,
    float4 const* bind_pose_inverse_bind_pose
) {
    float4 acc[4];

    float4 cm = centers_of_masses[i];
    float4 cm_0 = bind_pose_centers_of_masses[i];

    calculate_A_i(acc, masses[i], predicted_orientations[i], sizes[i], predicted_positions[i], bind_pose[i], cm, cm_0);

    unsigned base = i * N;
    for(unsigned ni = 0; ni < N; ni++) {
        float4 temp[4];
        float w = adjacency[base + ni];

        calculate_A_i(
            temp,
            masses[ni], predicted_orientations[ni], sizes[ni],
            predicted_positions[ni], bind_pose[ni],
            cm, cm_0
        );

        acc[0] = acc[0] + w * temp[0];
        acc[1] = acc[1] + w * temp[1];
        acc[2] = acc[2] + w * temp[2];
        acc[3] = acc[3] + w * temp[3];
    }

    float4 invRest[4];
    invRest[0] = bind_pose_inverse_bind_pose[i * 4 + 0];
    invRest[1] = bind_pose_inverse_bind_pose[i * 4 + 1];
    invRest[2] = bind_pose_inverse_bind_pose[i * 4 + 2];
    invRest[3] = bind_pose_inverse_bind_pose[i * 4 + 3];
    mat_mul(A, acc, invRest);
}

__global__ void k_calculate_particle_masses(unsigned N, float* d_masses, float4 const* d_sizes, float const* d_densities) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N) {
        float d_i = d_densities[i];
        float4 s_i = d_sizes[i];
        d_masses[i] = (4.0f / 3.0f) * glm::pi<float>() * s_i.x * s_i.y * s_i.z * d_i;
    }
}

__global__ void k_calculate_cluster_moment_matrices(
        float4* out, unsigned N,
        float const* adjacency,
        float const* masses,
        float4 const* predicted_orientations,
        float4 const* sizes,
        float4 const* predicted_positions,
        float4 const* bind_pose,
        float4 const* centers_of_masses,
        float4 const* bind_pose_centers_of_masses,
        float4 const* bind_pose_inverse_bind_pose) {
    unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
    if(id >= N) {
        return;
    }

    calculate_cluster_moment_matrix(
            &out[4 * id], id,
            adjacency, N,
            masses, predicted_orientations, sizes,
            predicted_positions, bind_pose,
            centers_of_masses, bind_pose_centers_of_masses,
            bind_pose_inverse_bind_pose
    );
}

__global__ void k_extract_rotations(
        float4* out, unsigned N,
        float4 const* A, float4 const* predicted_orientations
        ) {
    unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
    if(id >= N) {
        return;
    }

    out[id] = mueller_rotation_extraction_impl(&A[id * 4], predicted_orientations[id]);
}

template<long threads_per_block>
static long get_block_count(long N) {
    if(N < threads_per_block) {
        return 1;
    } else {
        return N / threads_per_block;
    }
}

class Compute_CUDA : public ICompute_Backend {
public:
    Compute_CUDA(hipStream_t stream)
    : stream(stream), current_particle_count(0) {
        printf("sb: CUDA compute backend created\n");
        // TODO(danielm): not sure if hipEventBlockingSync would be a good idea for this event
        ASSERT_CUDA_SUCCEEDED(hipEventCreateWithFlags(&ev_h_masses_ready, hipEventDefault));

        compute_ref = Make_Reference_Backend();
    }

    ~Compute_CUDA() override {
        hipEventDestroy(ev_h_masses_ready);
        hipStreamDestroy(stream);
    }

    size_t particle_count(System_State const& s) const {
        return s.position.size();
    }

#define SIZE_N_VEC1(N) ((N) *  1 * sizeof(float))
#define SIZE_N_VEC4(N) ((N) *  4 * sizeof(float))
#define SIZE_N_MAT4(N) ((N) * 16 * sizeof(float))

    void begin_new_frame(System_State const& s) override {
        hipError_t hr;
        auto N = particle_count(s);
        assert(N > 0);
        d_masses = CUDA_Array<float>(N);
        d_densities = CUDA_Array<float>(N);
        d_sizes = CUDA_Array<float4>(N);
        d_predicted_orientations = CUDA_Array<float4>(N);
        d_centers_of_masses = CUDA_Array<float4>(N);
        d_predicted_positions = CUDA_Array<float4>(N);
        d_bind_pose = CUDA_Array<float4>(N);
        d_bind_pose_centers_of_masses = CUDA_Array<float4>(N);
        d_bind_pose_inverse_bind_pose = CUDA_Array<float4>(4 * N);
        d_out = CUDA_Array<float4>(N);

        calculate_particle_masses(s);
        if(N != current_particle_count) {
            d_adjacency = CUDA_Array<float>(N * N);
            make_adjacency_matrix(s);
        }

        current_particle_count = N;
    }

    void calculate_particle_masses(System_State const& s) {
        auto const N = particle_count(s);

        h_masses.resize(N);

#define P_MASS_THREADS_PER_BLOCK (8)
        auto blocks = get_block_count<P_MASS_THREADS_PER_BLOCK>(N);
        hipMemcpyAsync(d_densities, s.density.data(), d_densities.bytes(), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_sizes, s.size.data(), d_sizes.bytes(), hipMemcpyHostToDevice, stream);
        k_calculate_particle_masses<<<blocks, P_MASS_THREADS_PER_BLOCK, 0, stream>>>(N, d_masses, d_sizes, d_densities);
        assert(h_masses.size() * sizeof(float) == d_masses.bytes());
        hipMemcpyAsync(h_masses.data(), d_masses, d_masses.bytes(), hipMemcpyDeviceToHost, stream);
        hipEventRecord(ev_h_masses_ready, stream);
    }

    void make_adjacency_matrix(System_State const& s) {
        DECLARE_BENCHMARK_BLOCK();

        BEGIN_BENCHMARK();
        auto const N = particle_count(s);
        Vector<float> h_ret;

        // TODO(danielm): are we sure that this fills the vector with 0.0f values?
        h_ret.resize(N * N);

        for(index_t i = 0; i < N; i++) {
            float* i_row = h_ret.data() + i * N;
            auto const& neighbors = s.edges.at(i);
            for(auto neighbor : neighbors) {
                i_row[neighbor] = 1.0f;
            }
        }

        hipMemcpyAsync(d_adjacency, h_ret.data(), d_adjacency.bytes(), hipMemcpyHostToDevice, stream);

        END_BENCHMARK();
        PRINT_BENCHMARK_RESULT_MASKED(0xF);
    }

    float mass_of_particle(System_State const&, index_t i) const {
        return h_masses[i];
    }

    void do_one_iteration_of_fixed_constraint_resolution(System_State& s, float phdt) override {
        compute_ref->do_one_iteration_of_fixed_constraint_resolution(s, phdt);
    }

    void do_one_iteration_of_distance_constraint_resolution(System_State& s, float phdt) override {
        compute_ref->do_one_iteration_of_distance_constraint_resolution(s, phdt);
    }

    void do_one_iteration_of_shape_matching_constraint_resolution(System_State& s, float dt) override {
        DECLARE_BENCHMARK_BLOCK();
        BEGIN_BENCHMARK();
        auto const N = particle_count(s);
        Vector<float4> h_centers_of_masses;
        h_centers_of_masses.reserve(N);

        ASSERT_CUDA_SUCCEEDED(d_predicted_orientations.write_async((float4*)s.predicted_orientation.data(), stream));
        ASSERT_CUDA_SUCCEEDED(d_predicted_positions.write_async((float4*)s.predicted_position.data(), stream));
        ASSERT_CUDA_SUCCEEDED(d_bind_pose.write_async((float4*)s.bind_pose.data(), stream));
        ASSERT_CUDA_SUCCEEDED(d_bind_pose_centers_of_masses.write_async((float4*)s.bind_pose_center_of_mass.data(), stream));
        ASSERT_CUDA_SUCCEEDED(d_bind_pose_inverse_bind_pose.write_async((float4*)s.bind_pose_inverse_bind_pose.data(), stream));

        hipEventSynchronize(ev_h_masses_ready);

        for (index_t i = 0; i < N; i++) {
            std::array<index_t, 1> me { i };
            auto& neighbors = s.edges[i];
            // TODO(danielm): C++14
            auto neighbors_and_me = iterator_union<decltype(neighbors.begin()), decltype(me.begin())>
                (neighbors.begin(), neighbors.end(), me.begin(), me.end());

            // Sum particle weights in the current cluster
            auto M = std::accumulate(
                neighbors.begin(), neighbors.end(),
                mass_of_particle(s, i),
                [&](float acc, index_t idx) {
                    return acc + mass_of_particle(s, idx);
                }
            );

            assert(M != 0);

            auto invRest = s.bind_pose_inverse_bind_pose[i];
            auto com0 = s.bind_pose_center_of_mass[i];

            // Center of mass calculated using the predicted positions
            auto com_cur = std::accumulate(
                neighbors.begin(), neighbors.end(),
                mass_of_particle(s, i) * s.predicted_position[i],
                [&](Vec4 const& acc, index_t idx) {
                    return acc + mass_of_particle(s, idx) * s.predicted_position[idx];
                }
            ) / M;

            s.center_of_mass[i] = com_cur;

            // TODO(danielm): check if the component order is correct
            h_centers_of_masses.push_back(make_float4(com_cur.x, com_cur.y, com_cur.z, com_cur.w));
        }

        ASSERT_CUDA_SUCCEEDED(d_centers_of_masses.write_async((float4*)h_centers_of_masses.data(), stream));

        Vector<Quat> h_out;
        h_out.resize(N);

        float4* d_tmp_cluster_moment_matrices = NULL;
        ASSERT_CUDA_SUCCEEDED(hipMalloc(&d_tmp_cluster_moment_matrices, N * 16 * sizeof(float)));

#define SHPMTCH_THREADS_PER_BLOCK (8)
        auto const blocks = get_block_count<SHPMTCH_THREADS_PER_BLOCK>(N);

        k_calculate_cluster_moment_matrices<<<blocks, SHPMTCH_THREADS_PER_BLOCK, 0, stream>>>(
            d_tmp_cluster_moment_matrices, N, d_adjacency, d_masses, d_predicted_orientations,
            d_sizes, d_predicted_positions, d_bind_pose, d_centers_of_masses,
            d_bind_pose_centers_of_masses, d_bind_pose_inverse_bind_pose
        );

        k_extract_rotations<<<blocks, SHPMTCH_THREADS_PER_BLOCK, 0, stream>>>(
            d_out, N,
            d_tmp_cluster_moment_matrices, d_predicted_orientations
        );

        hipFree(d_tmp_cluster_moment_matrices);
        
        // Calculate what we can on the CPU while we're waiting for the GPU

        struct Particle_Correction_Info {
            Vec4 pos_bind;
            Vec4 com_cur;
            float inv_numClusters;
        };

        Vector<Particle_Correction_Info> correction_infos;
        correction_infos.reserve(N);

        // Calculate what we can while we wait for the extracted quaternions
        // TODO(danielm): We could run this in a parallel cudaStream
        for (unsigned i = 0; i < N; i++) {
            auto const com0 = s.bind_pose_center_of_mass[i];
            correction_infos.push_back({});
            auto& inf = correction_infos.back();
            inf.pos_bind = s.bind_pose[i] - com0;
            inf.com_cur = s.center_of_mass[i];
            auto numClusters = NUMBER_OF_CLUSTERS(i);
            inf.inv_numClusters = 1.0f / (float)numClusters;
        }

        ASSERT_CUDA_SUCCEEDED(d_out.read_async((float4*)h_out.data(), stream));
        hipStreamSynchronize(stream);

        for (index_t i = 0; i < N; i++) {
            float const stiffness = 1;
            auto const R = h_out[i];

            auto& inf = correction_infos[i];

            // Rotate the bind pose position relative to the CoM
            auto pos_bind_rot = R * inf.pos_bind;
            // Our goal position
            auto goal = inf.com_cur + pos_bind_rot;
            // Number of clusters this particle is a member of
            auto correction = (goal - s.predicted_position[i]) * stiffness;
            // The correction must be divided by the number of clusters this particle is a member of
            s.predicted_position[i] += inf.inv_numClusters * correction;
            s.goal_position[i] = goal;
            s.predicted_orientation[i] = R;
        }

        END_BENCHMARK();
        PRINT_BENCHMARK_RESULT_MASKED(0xFF);
    }

private:
    hipStream_t stream;

    sb::Unique_Ptr<ICompute_Backend> compute_ref;

    hipEvent_t ev_h_masses_ready;
    Vector<float> h_masses;

    size_t current_particle_count;

    CUDA_Array<float> d_adjacency;
    CUDA_Array<float> d_masses;
    CUDA_Array<float> d_densities;
    CUDA_Array<float4> d_sizes;
    CUDA_Array<float4> d_predicted_orientations;
    CUDA_Array<float4> d_centers_of_masses;
    CUDA_Array<float4> d_predicted_positions;
    CUDA_Array<float4> d_bind_pose;
    CUDA_Array<float4> d_bind_pose_centers_of_masses;
    CUDA_Array<float4> d_bind_pose_inverse_bind_pose; // mat4x4
    CUDA_Array<float4> d_out;
};

static bool enumerate_devices() {
    hipError_t hr;
    int dev_count;

    printf("CUDA version: %d\n", CUDA_VERSION);

    if(CUDA_SUCCEEDED(hr, hipGetDeviceCount(&dev_count))) {
        hipDeviceProp_t prop;
        int dev_count_ok = 0;

        for(int i = 0; i < dev_count; i++) {
            if(CUDA_SUCCEEDED(hr, hipGetDeviceProperties(&prop, i))) {
                printf("Device #%d: '%s'\n", i, prop.name);
                dev_count_ok++;
            } else {
                printf("sb: failed to get properties of CUDA device #%d: hr=%d\n", i, hr);
            }
        }

        return dev_count_ok > 0;
    } else {
        printf("sb: failed to get CUDA device count: hr=%d\n", hr);
    }

    return false;
}

sb::Unique_Ptr<ICompute_Backend> Make_CUDA_Backend() {
    hipError_t hr;
    hipStream_t stream;

    if(enumerate_devices()) {
        if(CUDA_SUCCEEDED(hr, hipStreamCreate(&stream))) {
            auto ret = std::make_unique<Compute_CUDA>(stream);
            return ret;
        } else {
            printf("sb: failed to create CUDA stream: err=%d\n", hr);
        }
    }

    fprintf(stderr, "sb: can't make CUDA compute backend\n");
    return nullptr;
}
